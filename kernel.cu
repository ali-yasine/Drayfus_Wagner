#include "hip/hip_runtime.h"
//#include "Util.nqdsh"
//#include "csr.h"
#include "stdio.h"
#include "common.h"
#include "subsets.h"
#include "timer.h"
#include <stdlib.h>
#define MAX_THREADS 1024



static void handleSingletons(unsigned int* DP, unsigned int* apsp, unsigned int* allSubsets , unsigned int numTerminals, unsigned int num_nodes, unsigned int* terminals) {

    unsigned int totalSubsetCount = (1 << numTerminals) - 1;

    for(unsigned int vertex = 0; vertex < num_nodes; ++vertex) {
        for(unsigned int subset = 0; subset < numTerminals; ++subset) {
            
            //find index of 1 in subset
            unsigned int index = 0;
            for(unsigned int i = 0; i < numTerminals; ++i){
                if (allSubsets[subset * numTerminals + i]){
                    index = i;
                    break;
                }
            }
            DP[vertex * totalSubsetCount + subset] = apsp[vertex * num_nodes + terminals[index]];   
        }
    }
}

__global__ void DW_kernel(CsrGraph* graph, unsigned int numTerminals, unsigned int* terminals, unsigned int* DP, unsigned int* apsp, unsigned int* allSubsets, unsigned int numSubsets, unsigned int coarseFactor, unsigned int k, unsigned int subsetsDoneSoFar, unsigned int* subSubsets) {

    unsigned int root = blockIdx.x;
    unsigned int* subset = allSubsets + ((subsetsDoneSoFar + blockIdx.y ) * numTerminals);
    
    unsigned int num_sub_subsets = (1 << k) - 1;
    unsigned int* subSubets = subSubsets + num_sub_subsets * numTerminals * blockIdx.y;
    if ( threadIdx.x == 0) {
        generateSubsetsGPU(subset, numTerminals, subSubets);
    }
    __syncthreads();
    unsigned int* sMinusSS;
    hipMalloc(&sMinusSS, numTerminals * sizeof(unsigned int));
    if (root < graph->num_nodes && (blockIdx.y + subsetsDoneSoFar) * coarseFactor < numSubsets) {

        for(unsigned int sub_sub_set = threadIdx.x * coarseFactor; sub_sub_set < threadIdx.x * coarseFactor + coarseFactor ; ++sub_sub_set) {    
            if (sub_sub_set < num_sub_subsets) {

                unsigned int* subSubset = subSubets + (sub_sub_set * numTerminals);
                
                if (!equals(subset, subSubset, numTerminals)) {

                    unsigned int ss_index = getSubsetIndex(subSubset, numTerminals, allSubsets);
                    
                    setDifferenceGPU(subset, subSubset, numTerminals, sMinusSS);
                    
                    unsigned int sMinusSS_index = getSubsetIndex(sMinusSS, numTerminals, allSubsets);

                    for(unsigned int vertex = 0; vertex < graph->num_nodes; ++vertex) {

                        unsigned int v_to_sub_Subset = DP[vertex * numSubsets + ss_index];
                        unsigned int v_S_minusSS = DP[vertex * numSubsets + sMinusSS_index]; 
                        unsigned int root_to_v = apsp[root * graph->num_nodes + vertex]; 
                       
                        if (v_to_sub_Subset != UINT_MAX && v_S_minusSS != UINT_MAX && root_to_v != UINT_MAX) {

                            unsigned int sum = v_to_sub_Subset + v_S_minusSS + root_to_v;
                           
                            atomicMin(& DP[root * numSubsets + blockIdx.y + subsetsDoneSoFar], sum);
                        }   
                    }
                }
            }
        }
    }
    hipFree(sMinusSS);
}


void DrayfusWagnerGPU(CsrGraph* graph_cpu, CsrGraph* graph, unsigned int numTerminals, unsigned int* terminals, unsigned int* DP, unsigned int* apsp) {

    Timer timer; hipError_t err;

    unsigned int* allSubsets = getSortedSubsets(numTerminals);

    unsigned int *DP_d, *apsp_d, *allSubsets_d, *terminals_d;
    unsigned int numSubsets = (1 << numTerminals) - 1;

    for(unsigned int i = 0; i < graph_cpu->num_nodes * numSubsets; ++i)
      DP[i] = UINT_MAX;

    handleSingletons(DP, apsp, allSubsets, numTerminals, graph_cpu->num_nodes, terminals);
    startTime(&timer);


    //allocate memory 
    hipMalloc((void**) &DP_d, sizeof(unsigned int) * graph_cpu->num_nodes * numSubsets);
    hipMalloc((void**) &apsp_d, sizeof(unsigned int) * graph_cpu->num_nodes * graph_cpu->num_nodes);
    hipMalloc((void**) &allSubsets_d, sizeof(unsigned int) * numSubsets * numTerminals);
    hipMalloc((void**) &terminals_d, sizeof(unsigned int) * numTerminals);
    
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Allocation Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time: ");
    
    startTime(&timer);
    //copy data to device
    hipMemcpy(DP_d, DP, sizeof(unsigned int) * graph_cpu->num_nodes * numSubsets, hipMemcpyHostToDevice);
    hipMemcpy(apsp_d, apsp, sizeof(unsigned int) * graph_cpu->num_nodes * graph_cpu->num_nodes, hipMemcpyHostToDevice);
    hipMemcpy(allSubsets_d, allSubsets, sizeof(unsigned int) * numSubsets * numTerminals, hipMemcpyHostToDevice);
    hipMemcpy(terminals_d, terminals, sizeof(unsigned int) * numTerminals, hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Copy Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    stopTime(&timer);

    printElapsedTime(timer, "Copy to device time: ");

    unsigned int subsetsDoneSoFar = numTerminals;

    startTime(&timer);
    //launch kernel
    for(unsigned int k = 2; k <= numTerminals; k++) {

        unsigned int numThreads = MAX_THREADS;
        unsigned int coarseFactor = 1;
        unsigned int currSubsetNum = choose(numTerminals, k);
        unsigned int subSubetsNum = (1 << k) - 1;
        
        if (MAX_THREADS < subSubetsNum) 
            coarseFactor = (MAX_THREADS +  subSubetsNum - 1) / subSubetsNum;
        
        else 
            numThreads = (1 << k) - 1;

        unsigned int* subSubets;
        hipMalloc((void**) &subSubets, ((1 << k) - 1) * currSubsetNum * numTerminals * sizeof(unsigned int));
        
        err = hipGetLastError();
        if (err != hipSuccess) 
            printf("subSubets malloc Error: %s in launch num: %u and subsetsSoFar: %u\n", hipGetErrorString(err), k - 1, subsetsDoneSoFar);
        
        hipDeviceSynchronize();

        dim3 numBlocks (graph_cpu->num_nodes, currSubsetNum);

        DW_kernel<<<numBlocks, numThreads>>>(graph, numTerminals, terminals_d, DP_d, apsp_d, allSubsets_d, numSubsets, coarseFactor, k, subsetsDoneSoFar, subSubets);
        
        err = hipGetLastError();
        if (err != hipSuccess) 
            printf("Kernel Error: %s in launch num: %u and subsetsSoFar: %u\n", hipGetErrorString(err), k - 1, subsetsDoneSoFar);
            
        hipFree(subSubets);
        hipDeviceSynchronize();
        subsetsDoneSoFar += currSubsetNum;
    }
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time: ", GREEN);

    //copy data back to host
    startTime(&timer);
    
    hipMemcpy(DP, DP_d, sizeof(unsigned int) * graph_cpu->num_nodes * numSubsets, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Copy Back Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    stopTime(&timer);

    printElapsedTime(timer, "Copy to host time: ");
    
    //free memory
    hipFree(DP_d);
    hipFree(apsp_d);
    hipFree(allSubsets_d);
    hipFree(terminals_d);
    hipDeviceSynchronize();

    free(allSubsets);
    
}
