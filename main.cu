#include <assert.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

#include "DrayfusWagner.h"
#include "timer.h"
#include "common.h"
#include "Coo.h"
void verify(unsigned int * DP, unsigned int * DP_d, unsigned int num_nodes, unsigned int num_subsets, unsigned int* allSubsets, unsigned int numberOfTerminals){
  unsigned int num_mismatch = 0;
  for(unsigned int v = 0; v < num_nodes; ++v) {
    for (unsigned int subset = 0; subset < num_subsets; ++subset) {
      if (DP[v * num_subsets + subset] != DP_d[v * num_subsets + subset]) {
        num_mismatch++;
        if (num_mismatch < 10)
          printf("mismatch at vertex v: %u,subset: %u DP: %u\tDP_d: %u\n", v, subset, DP[v * num_subsets + subset], DP_d[v * num_subsets + subset]);
      }
    }
  }
  if( num_mismatch > 0) {
    printf("Number of mismatches: %u\n", num_mismatch);
  }
}

void verifyFlippedDP(unsigned int* DP, unsigned int* Dp_d, unsigned int num_nodes, unsigned int num_subsets) {
  unsigned int num_mismatch = 0;
  for(unsigned int v = 0; v < num_nodes; ++v) {
    for (unsigned int subset = 0; subset < num_subsets; ++subset) {
      if (DP[v * num_subsets + subset] != Dp_d[subset * num_nodes + v]) {
        num_mismatch++;
        if (num_mismatch < 10)
          printf("mismatch at vertex v: %u\t, subset: %u \tDP: %u\tDP_d: %u\n", v, subset, DP[v * num_subsets + subset], Dp_d[subset * num_nodes + v]);
      }
    }
  } 
  if( num_mismatch > 0) 
    printf("Number of mismatches: %u\n", num_mismatch);
}

int main(int argc, char** argv) {
  hipDeviceSynchronize();
  setbuf(stdout, NULL);
  
  unsigned int num_nodes = (argc > 1) ?  atoi(argv[1]) : 300;
  unsigned int numberOfTerminals = (argc > 2) ?  atoi(argv[2]) : 8;
  
  unsigned int* terminals = (unsigned int*) malloc(sizeof(unsigned int) * numberOfTerminals);

  for (unsigned int i = 0; i < numberOfTerminals; ++i) {
    terminals[i] = rand() % num_nodes;
  }
  generateCOOGraph(num_nodes);

  
  char filename[100];
  sprintf(filename, "data/%u.txt", num_nodes);
  CsrGraph* graph = readCSRgraph(filename);
 
  Timer timer;
  
  startTime(&timer);
  printf("Computing Floyd-Warshall...\n");
  unsigned int* apsp = floydWarshall(*graph);
  stopTime(&timer);
  printElapsedTime(timer, "Floyd-Warshall");
  
  startTime(&timer);
  
  printf("Running CPU version\n");

  unsigned int* cpuResult = DrayfusWagner_cpu(*graph, numberOfTerminals, terminals, apsp);
  
  stopTime(&timer);
  printElapsedTime(timer, "CPU time", CYAN);
  
  printf("Running GPU version\n");
  // startTime(&timer);
  // Allocate GPU memory
  CsrGraph* graph_d = createEmptyCSRGraphOnGPU(graph->num_nodes, graph->num_edges);

  // Copy graph to GPU

  copyCSRGraphToGPU(graph, graph_d);
  hipDeviceSynchronize();

  unsigned int* DP = (unsigned int*) malloc(sizeof(unsigned int) * graph->num_nodes *  ((1 << numberOfTerminals) - 1) );
  
  
  startTime(&timer);
  DrayfusWagnerGPU(graph, graph_d, numberOfTerminals, terminals, DP, apsp);
  stopTime(&timer);

  printElapsedTime(timer, "GPU total time", CYAN);

  verify(cpuResult, DP, graph->num_nodes , ((1 << numberOfTerminals) - 1), getSortedSubsets(numberOfTerminals), numberOfTerminals);
  free(DP);

  CsrGraph* graph_opt1_d = createEmptyCSRGraphOnGPU(graph->num_nodes, graph->num_edges);
  copyCSRGraphToGPU(graph, graph_opt1_d);
  hipDeviceSynchronize();



  unsigned int* DP_opt1 = (unsigned int*) malloc(sizeof(unsigned int) * graph->num_nodes *  ((1 << numberOfTerminals) - 1) );
  startTime(&timer);
  DrayfusWagnerGPU_o1(graph, graph_opt1_d, numberOfTerminals, terminals, DP_opt1, apsp);
  stopTime(&timer);
  printElapsedTime(timer, "GPU opt1 time", CYAN);
  verifyFlippedDP(cpuResult, DP_opt1, graph->num_nodes, ((1 << numberOfTerminals) - 1));
  free(DP_opt1);

  CsrGraph* graph_opt2_d = createEmptyCSRGraphOnGPU(graph->num_nodes, graph->num_edges);
  copyCSRGraphToGPU(graph, graph_opt2_d);
  hipDeviceSynchronize(); 
  unsigned int* DP_opt2 = (unsigned int*) malloc(sizeof(unsigned int) * graph->num_nodes *  ((1 << numberOfTerminals) - 1) );
  
  startTime(&timer);
  DrayfusWagnerGPU_o2(graph, graph_opt2_d, numberOfTerminals, terminals, DP_opt2, apsp);
  stopTime(&timer);
  printElapsedTime(timer, "GPU opt2 time", CYAN);
  verifyFlippedDP(cpuResult, DP_opt2, graph->num_nodes, ((1 << numberOfTerminals) - 1));


  free(DP_opt2);
  free(apsp);
  free(cpuResult);
}
