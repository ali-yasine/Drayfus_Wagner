#include "csr.h"

#include <cstdlib>
#include <stdio.h>
#include <assert.h>

CsrGraph* createEmptyCSRGraphOnGPU(unsigned int num_nodes, unsigned int num_edges) {

    CsrGraph graph_shadow;
    graph_shadow.num_nodes = num_nodes;
    graph_shadow.num_edges = num_edges;
    hipMalloc((void**) &graph_shadow.row_offsets, sizeof(unsigned int) * (num_nodes + 1));
    hipMalloc((void**) &graph_shadow.col_indices, sizeof(unsigned int) * num_edges);
    hipMalloc((void**) &graph_shadow.edge_weights, sizeof(unsigned int) * num_edges);

    CsrGraph* graph;

    hipMalloc((void**) &graph, sizeof(CsrGraph));
    hipMemcpy(graph, &graph_shadow, sizeof(CsrGraph), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    return graph;
}

void freeCSRGraphOnGPU(CsrGraph* graph) {
    
    hipFree(graph->row_offsets);
    hipFree(graph->col_indices);
    hipFree(graph->edge_weights);
    hipFree(graph);
}
void copyCSRGraphToGPU(CsrGraph* graph, CsrGraph* graph_d) {
    CsrGraph graph_shadow;
    hipMemcpy(&graph_shadow, graph_d, sizeof(CsrGraph), hipMemcpyDeviceToHost);
    
    assert(graph_shadow.num_nodes == graph->num_nodes);
    assert(graph_shadow.num_edges == graph->num_edges);

    hipMemcpy(graph_shadow.row_offsets, graph->row_offsets, (graph->num_nodes + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(graph_shadow.col_indices, graph->col_indices, graph->num_edges * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(graph_shadow.edge_weights, graph->edge_weights, graph->num_edges * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}